#include "hip/hip_runtime.h"
/**
 * @file gpu_fma.cu
 * @author UCHINO Yuki
 * @brief This function computes D = fma(A,B,C).
 * @version 1.0
 * @date 2024-02-23
 * @copyright Copyright (c) 2024 UCHINO Yuki
**/

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

void __global__ gpuFMA(double const * const A,
        double const * const B,
        double const * const C,
        double * const D,
        int const N){
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < N){
        D[i] = fma(A[i],B[i],C[i]);
    }
}

// D = fma(A,B,C)
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare all variables.*/
    mxGPUArray const *A;
    mxGPUArray const *B;
    mxGPUArray const *C;
    mxGPUArray *D;

    /* Choose a reasonably sized number of threads for the block. */
    int const threadsPerBlock = 1024;
    int blocksPerGrid;

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();

    A = mxGPUCreateFromMxArray(prhs[0]);
    B = mxGPUCreateFromMxArray(prhs[1]);
    C = mxGPUCreateFromMxArray(prhs[2]);

    int N;
    double const *d_A;
    double const *d_B;
    double const *d_C;
    double *d_D;

    /*
    * Now that we have verified the data type, extract a pointer to the input
    * data on the device.
    */
    d_A = (double const *)(mxGPUGetDataReadOnly(A));
    d_B = (double const *)(mxGPUGetDataReadOnly(B));
    d_C = (double const *)(mxGPUGetDataReadOnly(C));

    /* Create a GPUArray to hold the result and get its underlying pointer. */
    D = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(A),
                            mxGPUGetDimensions(A),
                            mxGPUGetClassID(A),
                            mxGPUGetComplexity(A),
                            MX_GPU_DO_NOT_INITIALIZE);
    d_D = (double *)(mxGPUGetData(D));

    /*
    * Call the kernel using the CUDA runtime API. We are using a 1-d grid here,
    * and it would be possible for the number of elements to be too large for
    * the grid. For this example we are not guarding against this possibility.
    */
    N = (int)(mxGPUGetNumberOfElements(A));
    blocksPerGrid = (N + 1023) >> 10;
    gpuFMA<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, d_D, N);

    /* Wrap the result up as a MATLAB gpuArray for return. */
    plhs[0] = mxGPUCreateMxArrayOnGPU(D);

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    mxGPUDestroyGPUArray(A);
    mxGPUDestroyGPUArray(B);
    mxGPUDestroyGPUArray(C);
    mxGPUDestroyGPUArray(D);
}